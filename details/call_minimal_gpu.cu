#include "call_minimal_gpu.h"

#include <algorithm>
#include <cassert>
#include <cstddef>
#include <cstdint>
#include <cstdlib>
#include <iostream>
#include <iomanip>
#include <cmath>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "minimal.cuh"

extern __host__ void call_minimal_gpu( 
    double const* __restrict__ a_begin, 
    double const* __restrict__ a_end, 
    double const* __restrict__ b_begin, 
    double* __restrict__ c_begin, 
    int const num_of_blocks, int const threads_per_block );


void __host__ call_minimal_gpu( 
    double const* __restrict__ a_begin, 
    double const* __restrict__ a_end, 
    double const* __restrict__ b_begin, 
    double* __restrict__ c_begin, 
    int const num_of_blocks, 
    int const threads_per_block )
{
    if( ( a_begin != nullptr ) && ( a_end   != nullptr ) &&
        ( b_begin != nullptr ) && ( c_begin != nullptr ) )
    {
        std::ptrdiff_t const temp = std::distance( a_begin, a_end );
        
        if( temp > 0 )
        {
            uint64_t const N = temp;
            std::size_t const N_SIZE = sizeof( double ) * N;
            
            double* cuda_a = nullptr;
            double* cuda_b = nullptr;
            double* cuda_c = nullptr;
            
            ::hipMalloc( &cuda_a, N_SIZE );
            ::hipMalloc( &cuda_b, N_SIZE );
            ::hipMalloc( &cuda_c, N_SIZE );
            
            ::hipMemcpy( cuda_a, a_begin, N_SIZE, hipMemcpyHostToDevice );
            ::hipMemcpy( cuda_b, b_begin, N_SIZE, hipMemcpyHostToDevice );
            ::hipMemcpy( cuda_c, c_begin, N_SIZE, hipMemcpyHostToDevice );
            
            int device = 0;
            ::hipGetDevice( &device );
            
            ::hipDeviceProp_t deviceProp;
            ::hipGetDeviceProperties( &deviceProp, device);
            std::cout << deviceProp.major << "." 
                      << deviceProp.minor << std::endl;
                    
            ::hipDeviceSynchronize();
            hipError_t err = ::hipGetLastError();
            assert( err == hipSuccess );
            
            calculateSum<<< num_of_blocks, threads_per_block >>>( 
                N, cuda_a, cuda_b, cuda_c );
            
            err = ::hipPeekAtLastError();
            assert( err == ::hipSuccess );
            
            ::hipMemcpy( c_begin, cuda_c, N_SIZE, hipMemcpyDeviceToHost );
            
            ::hipFree( cuda_a );
            ::hipFree( cuda_b );
            ::hipFree( cuda_c );
        }        
    }
    
    return;
}
