#include <cassert>
#include <cstddef>
#include <cstdint>
#include <cstdlib>
#include <iostream>
#include <iomanip>
#include <cmath>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "impl/minimal.cuh"

int main()
{
    uint64_t const N = 1000;
    std::size_t const N_SIZE = sizeof( double ) * N;
    
    std::vector< double > A( N, 1.0 );
    std::vector< double > B( N, 2.0 );
    std::vector< double > C( N, 0.0 );
    
    double* cuda_a = nullptr;
    double* cuda_b = nullptr;
    double* cuda_c = nullptr;
    
    hipMalloc( &cuda_a, N_SIZE );
    hipMalloc( &cuda_b, N_SIZE );
    hipMalloc( &cuda_c, N_SIZE );
    
    hipMemcpy( cuda_a, A.data(), N_SIZE, hipMemcpyHostToDevice );
    hipMemcpy( cuda_b, B.data(), N_SIZE, hipMemcpyHostToDevice );
    hipMemcpy( cuda_c, C.data(), N_SIZE, hipMemcpyHostToDevice );
    
    int device = 0;
    hipGetDevice( &device );
    
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties( &deviceProp, device);
    std::cout << deviceProp.major << "." << deviceProp.minor << std::endl;
            
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    assert( err == hipSuccess );
    
    calculateSum<<< 64, 64 >>>( N, cuda_a, cuda_b, cuda_c );
    err = hipPeekAtLastError();
    assert( err == hipSuccess );
    
    hipMemcpy( C.data(), cuda_c, N_SIZE, hipMemcpyDeviceToHost );
    
    hipFree( cuda_a );
    hipFree( cuda_b );
    hipFree( cuda_c );
    
    return 0;
}

